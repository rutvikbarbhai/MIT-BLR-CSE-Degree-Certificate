#include <iostream>
#include <hip/hip_runtime.h>

#define N 10 // Number of elements

__global__ void oddEvenSortKernel(int *arr, int n, int phase) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 2 * idx + phase; // Compute index for even or odd phase

    if (i + 1 < n) {
        if (arr[i] > arr[i + 1]) {
            // Swap adjacent elements
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

void oddEvenSort(int *arr, int n) {
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / (2 * blockSize);

    // Perform n iterations for sorting
    for (int phase = 0; phase < n; phase++) {
        oddEvenSortKernel<<<numBlocks, blockSize>>>(d_arr, n, phase % 2);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int arr[N] = {64, 25, 12, 22, 11, 90, 35, 29, 78, 5};

    std::cout << "Original array: ";
    for (int i = 0; i < N; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;

    oddEvenSort(arr, N);

    std::cout << "Sorted array: ";
    for (int i = 0; i < N; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;

    return 0;
}
