#include <iostream>
#include <hip/hip_runtime.h>

__global__ void selectionSortKernel(int *arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        // Find the minimum element in the remaining unsorted array
        for (int j = idx + 1; j < n; j++) {
            if (arr[j] < arr[idx]) {
                // Swap if the found minimum is smaller
                int temp = arr[idx];
                arr[idx] = arr[j];
                arr[j] = temp;
            }
        }
    }
}

void selectionSort(int *arr, int n) {
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with enough threads for all elements
    int blockSize = 256; // Number of threads per block
    int numBlocks = (n + blockSize - 1) / blockSize;

    selectionSortKernel<<<numBlocks, blockSize>>>(d_arr, n);
    hipDeviceSynchronize();

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    const int n = 10;
    int arr[n] = {64, 25, 12, 22, 11, 90, 35, 29, 78, 5};

    std::cout << "Original array: ";
    for (int i = 0; i < n; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;

    selectionSort(arr, n);

    std::cout << "Sorted array: ";
    for (int i = 0; i < n; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;

    return 0;
}
