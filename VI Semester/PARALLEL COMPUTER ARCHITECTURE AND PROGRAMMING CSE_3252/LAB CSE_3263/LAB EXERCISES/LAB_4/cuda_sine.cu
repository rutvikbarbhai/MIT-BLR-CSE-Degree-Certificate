#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void calculateSine(float *angles, float *sineResults, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        sineResults[i] = sin(angles[i]);
    }
}

int main() {
    int N = 5;
    int size = N * sizeof(float);
    float angles[] = {0.0, 0.5236, 1.5708, 2.0944, 3.1416}; // Angles in radians
    float sineResults[N];

    float *d_angles, *d_sineResults;

    hipMalloc((void **)&d_angles, size);
    hipMalloc((void **)&d_sineResults, size);

    hipMemcpy(d_angles, angles, size, hipMemcpyHostToDevice);

    calculateSine<<<1, N>>>(d_angles, d_sineResults, N);

    hipMemcpy(sineResults, d_sineResults, size, hipMemcpyDeviceToHost);

    printf("Sine results: ");
    for (int i = 0; i < N; i++) {
        printf("%.4f ", sineResults[i]);
    }
    printf("\n");

    hipFree(d_angles);
    hipFree(d_sineResults);

    return 0;
}


commands

sudo apt update
sudo apt install nvidia-cuda-toolkit
nvcc -o vectorAdd vectorAdd.cu
