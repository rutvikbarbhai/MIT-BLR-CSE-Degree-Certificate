#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *A, int *B, int *C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 5;
    int size = N * sizeof(int);
    int A[] = {1, 2, 3, 4, 5};
    int B[] = {5, 4, 3, 2, 1};
    int C[N];

    int *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    addVectors<<<1, N>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Resultant vector C: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", C[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
