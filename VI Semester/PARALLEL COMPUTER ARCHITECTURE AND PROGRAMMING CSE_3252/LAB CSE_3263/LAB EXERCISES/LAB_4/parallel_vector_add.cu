#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *A, int *B, int *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1024;
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    int size = N * sizeof(int);
    int A[N], B[N], C[N];

    // Initialize vectors A and B
    for (int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = N - i;
    }

    int *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    addVectors<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print first 10 elements of C
    for (int i = 0; i < 10; i++) {
        printf("%d ", C[i]);
    }
    printf("\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

